#include "hip/hip_runtime.h"
/*  
 *  uniform_cuda_kernel.cu
 *  
 *  Author(s):
 *  Matteo Spallanzani <spmatteo@iis.ee.ethz.ch>
 *  
 *  Copyright (c) 2020-2021 ETH Zurich. All rights reserved.
 *  
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *  
 *  http://www.apache.org/licenses/LICENSE-2.0
 *  
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */ 

#include <torch/extension.h>
#include <vector>

// #include <stdio.h>  // for debugging

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#define THREADS_PER_BLOCK 1024

#define PLUS_1(x) (x + 1)
#define ABS(x) ((x < 0.0f) ? -x : x)
#define CLAMP_0_1(x) ((x > 1.0f) ? 1.0f : ((x < 0.0f) ?  0.0f : x))


// definitions of CUDA kernels (executed on: GPU)


template <typename scalar_t>
__global__ void uniform_forward_cuda_kernel_pmf(
    scalar_t * const __restrict__ pmf,
    scalar_t * const __restrict__ x_in,
    const int64_t len_x,
    const scalar_t * __restrict__ t,      // thresholds
    const int64_t len_t,
    const scalar_t * __restrict__ mu,     // mean
    const scalar_t * __restrict__ sigma,  // standard deviation
    const scalar_t * __restrict__ training
)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;

    if (ix < len_x)
    {
        // pre-compute row offset from the beginning of the `pmf` array
        int row_offset = ix * PLUS_1(len_t);

        // compute shifted thresholds
        for (int it = 0; it < PLUS_1(len_t); ++it)
        {
            pmf[row_offset + it + 1] = x_in[ix] - *mu - t[it];
        }

        // compute CDF
        for (int it = 0; it < PLUS1(len_t); ++it)
        {
            if (it == 0)
            {
                pmf[row_offset + it] = 1.0f;
            }
            else
            {
                if (*training && (*sigma != 0.0f))
                {
                    scalar_t sigma_inv = 1.0 / (*sigma);
                    pmf[row_offset + it] = CLAMP_0_1(0.5f * (temp[row_offset + it] * sigma_inv + 1.0f));
                }
                else
                {
                    pmf[row_offset + it] = (scalar_t) (pmf[row_offset + it] >= 0.0f);
                }
            }
        }

        // compute the probability mass in each bin
        for (int iq = 0; iq < PLUS_1(len_t) - 1; ++iq)
        {
            pmf[row_offset + iq] = pmf[row_offset + iq] - pmf[row_offset + iq + 1];
        }
        // the last bin (with index `row_offset + len_t`) would have mass `pmf[row_offset + len_t] - 0.0f`, so it's not necessary to compute it!
    }
    else  // I am out of bounds!
    {
        return;
    }
}


template <typename scalar_t>
__global__ void uniform_forward_cuda_kernel_expectation(
    scalar_t * const __restrict__ x_out,
    scalar_t * const __restrict__ pmf,
    const int64_t len_x,
    const scalar_t * __restrict__ q,
    const int64_t len_t
)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;

    if (ix < len_x)
    {
        // pre-compute row offset from the beginning of the `pmf` array
        int row_offset = ix * PLUS_1(len_t);

        scalar_t sum = 0.0f;
        for (int iq = 0; iq < PLUS_1(len_t); ++iq)
        {
            sum += q[iq] * pmf[row_offset + iq];
        }

        x_out[ix] = sum;
    }
    else  // I am out of bounds!
    {
        return;
    }
}


template <typename scalar_t>
__global__ void uniform_forward_cuda_kernel_mode(
    scalar_t * const __restrict__ x_out,
    scalar_t * const __restrict__ pmf,
    const int64_t len_x,
    const scalar_t * __restrict__ q,
    const int64_t len_t
)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;

    if (ix < len_x)
    {
        // pre-compute row offset from the beginning of the `pmf` array
        int row_offset = ix * PLUS_1(len_t);

        // find the bin that contains the greatest probability mass
        int32_t argmax = 0;
        scalar_t max = pmf[row_offset + argmax];
        for (int iq = 1; iq < PLUS_1(len_t); ++iq)
        {
            if (max < temp[row_offset + iq])
            {
                argmax = iq;
                max = pmf[row_offset + argmax];
            }
        }

        x_out[ix] = q[argmax];
    }
    else  // I am out of bounds!
    {
        return;
    }
}


template <typename scalar_t>
__global__ void uniform_forward_cuda_kernel_random(
    scalar_t * const __restrict__ x_out,
    scalar_t * const __restrict__ us,     // samples from the uniform over [0, 1)
    scalar_t * const __restrict__ pmf,
    const int64_t len_x,
    const scalar_t * __restrict__ q,
    const int64_t len_t
)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;

    if (ix < len_x)
    {
        // pre-compute row offset from the beginning of the `pmf` array
        int row_offset = ix * PLUS_1(len_t);

        // Each row in `pmf` sums to 1 due to the normalisation property.
        // I imagine to have a segment for each bin, the length of the
        // segment being proportional to the probability mass in the bin. If I
        // glue the segments in a row, selecting a random number in [0, 1)
        // will generate a point falling in exactly one of the segments, i.e.,
        // in one of the bins.
        scalar_t u = us[ix];
        scalar_t cum_prob = 0.0f;
        int idx = -1;
        for (int iq = 0; iq < PLUS_1(iq); ++iq)
        {
            cum_prob += pmf[row_offset + iq];
            if ((idx < 0) && (u < cum_prob))  // I work under the assumption that the cumulative probability is monotone
            {
                idx = it;  // setting this integer to positive acts as a flag signaling that the sampled bin has been found
            }
        }

        x_out[ix] = q[idx];
    }
    else  // I am out of bounds!
    {
        return;
    }
}


template <typename scalar_t>
__global__ void uniform_backward_cuda_kernel(
    scalar_t * const __restrict__ grad_out,
    const scalar_t * __restrict__ grad_in,
    const scalar_t * __restrict__ x_in,
    const int64_t len_x,
    const scalar_t * __restrict__ q,
    const scalar_t * __restrict__ t,
    const int64_t len_t,
    const scalar_t * __restrict__ mu,
    const scalar_t * __restrict__ sigma
)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix < len_x)
    {
        scalar_t sum = 0.0f;

        for (int it = 0; it < len_t; ++it)
        {
            // input position relative to the threshold
            scalar_t x_minus_t = x_in[ix] - t[it] - *mu;

            // the derivative of the expected (i.e., regularised) step function is the PDF of the uniform distribution
            scalar_t pdf;
            if (*sigma != 0.0f)
            {
                scalar_t sigma_inv = 1.0f / (*sigma);
                scalar_t local_derivative = (scalar_t) (ABS(x_minus_t) <= (*sigma));
                pdf = 0.5f * sigma_inv * local_derivative;
            }
            else
            {
                pdf = 0.0f;  // no noise, no gradient!
            }

            // dilate and accumulate expected derivative
            scalar_t dq = q[it + 1] - q[it];
            sum += dq * pdf;
        }

        // compose gradients
        grad_out[ix] = sum * grad_in[ix];
    }
    else  // I am out of bounds!
    {
        return;
    }
}


// definitions of C++\CUDA interface (executed on: CPU)
// goals:
//   * allocate GPU memory for the output;
//   * define the parameters for the GPU kernel;
//   * call the kernel;

torch::Tensor uniform_forward_cuda_dispatch(
    torch::Tensor x_in,
    torch::Tensor q,
    torch::Tensor t,
    torch::Tensor mu,
    torch::Tensor sigma,
    torch::Tensor strategy,
    torch::Tensor training
)
{
    auto x_out = torch::zeros_like(x_in);
    auto pmf = torch::zeros({x_in.numel(), PLUS2(t.numel())}, torch::TensorOptions().dtype(x_in.dtype()).device(x_in.device()));

    const dim3 blocks((x_in.numel() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

    // compute PMF over bins (i.e., the quantization levels)
    AT_DISPATCH_FLOATING_TYPES(
        x_in.type(),
        "uniform_forward_cuda_kernel_pmf",
        ([&] {
            uniform_forward_cuda_kernel_pmf<scalar_t><<<blocks, THREADS_PER_BLOCK>>>(
                pmf.data_ptr<scalar_t>(),
                x_in.data_ptr<scalar_t>(),
                x_in.numel(),
                t.data_ptr<scalar_t>(),
                t.numel(),
                mu.data_ptr<scalar_t>(),
                sigma.data_ptr<scalar_t>(),
                training.data_ptr<scalar_t>()
            );
        })
    );

    switch(strategy.item())
    {
        case 0:  // expectation
            AT_DISPATCH_FLOATING_TYPES(
                x_in.type(),
                "uniform_forward_cuda_kernel_expectation",
                ([&] {
                    uniform_forward_cuda_kernel_expectation<scalar_t><<<blocks, THREADS_PER_BLOCK>>>(
                        x_out.data_ptr<scalar_t>(),
                        pmf.data_ptr<scalar_t>(),
                        x_in.numel(),
                        q.data_ptr<scalar_t>(),
                        t.numel()
                    );
                })
            );

        case 1:  // argmax sampling (i.e., mode)
            AT_DISPATCH_FLOATING_TYPES(
                x_in.type(),
                "uniform_forward_cuda_kernel_mode",
                ([&] {
                    uniform_forward_cuda_kernel_mode<scalar_t><<<blocks, THREADS_PER_BLOCK>>>(
                        x_out.data_ptr<scalar_t>(),
                        pmf.data_ptr<scalar_t>(),
                        x_in.numel(),
                        q.data_ptr<scalar_t>(),
                        t.numel()
                    );
                })
            );

        case 2:  // random sampling
            auto us = torch::rand_like(x_in);
            AT_DISPATCH_FLOATING_TYPES(
                x_in.type(),
                "uniform_forward_cuda_kernel_random",
                ([&] {
                    uniform_forward_cuda_kernel_random<scalar_t><<<blocks, THREADS_PER_BLOCK>>>(
                        x_out.data_ptr<scalar_t>(),
                        us.data_ptr<scalar_t>(),
                        pmf.data_ptr<scalar_t>(),
                        x_in.numel(),
                        q.data_ptr<scalar_t>(),
                        t.numel()
                    );
                })
            );

    }



    return x_out;
}


torch::Tensor uniform_backward_cuda_dispatch(
    torch::Tensor grad_in,
    torch::Tensor x_in,
    torch::Tensor q,
    torch::Tensor t,
    torch::Tensor mu,
    torch::Tensor sigma
)
{
    auto grad_out = torch::zeros_like(x_in);
    const dim3 blocks((x_in.numel() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

    // see PyTorch's ATen ("A TENsor") library; the full path from PyTorch GitHub repo's main is `aten/src/ATen/Dispatch.h`
    AT_DISPATCH_FLOATING_TYPES(
        x_in.type(),
        "uniform_backward_cuda",
        ([&] {
            uniform_backward_cuda_kernel<scalar_t><<<blocks, THREADS_PER_BLOCK>>>(
                grad_out.data_ptr<scalar_t>(),
                grad_in.data_ptr<scalar_t>(),
                x_in.data_ptr<scalar_t>(),
                x_in.numel(),
                q.data_ptr<scalar_t>(),
                t.data_ptr<scalar_t>(),
                t.numel(),
                mu.data_ptr<scalar_t>(),
                sigma.data_ptr<scalar_t>()
            );
        })
    );

    return grad_out;
}

